#include "hip/hip_runtime.h"
#include "process.hpp"
#include <stdio.h>

__global__ void sobel_xy(const uint8_t* in, uint8_t *out_x, uint8_t *out_y,
                            int width, int height, int pitchIn,
                            int pitchX, int pitchY) {
    int r = 1;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < r || x >= width - r) return;
    if (y < r || y >= height - r) return;

    int pix00 = in[(y - r) * pitchIn + x - r];
    int pix01 = in[(y - r) * pitchIn + x];
    int pix02 = in[(y - r) * pitchIn + x + r];
    int pix10 = in[y * pitchIn + x - r];
    int pix12 = in[y * pitchIn + x + r];
    int pix20 = in[(y + r) * pitchIn + x - r];
    int pix21 = in[(y + r) * pitchIn + x];
    int pix22 = in[(y + r) * pitchIn + x + r];

    int sumX = -pix00 + pix02 - 2*pix10 + 2*pix12 - pix20 + pix22;
    int sumY =  pix00 + 2*pix01 + pix02 - pix20 - 2*pix21 - pix22;

    out_x[x + y * pitchX] = (sumX > 0) ? sumX : -sumX;
    out_y[x + y * pitchY] = (sumY > 0) ? sumY : -sumY;
}

void sobel_filter(const uint8_t* devIn, uint8_t *devX, uint8_t *devY,
                    int width, int height, int pitchIn,
                    int pitchX, int pitchY) {

    int bsize_w = 32;
    int bsize_h = 16;
    int w     = std::ceil((float)width / bsize_w);
    int h     = std::ceil((float)height / bsize_h);

    dim3 dimBlock(bsize_w, bsize_h);
    dim3 dimGrid(w, h);

    sobel_xy<<<dimGrid, dimBlock>>>(devIn, devX, devY, width, height, pitchIn, pitchX, pitchY);
    hipDeviceSynchronize();

    if (hipPeekAtLastError())
        printf("sobel filter Error\n");
}


__global__ void compute_avg_pooling(const uint8_t* sobelx, const uint8_t* sobely,
                                    uint8_t *out, int patchs_x, int patchs_y,
                                    int pitchX, int pitchY, int pitchOut) {

    __shared__ int local_sum;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= patchs_x * POOLSIZE || y >= patchs_y * POOLSIZE) return;

    if(threadIdx.x == 0 && threadIdx.y == 0) local_sum = 0;
    __syncthreads();

    atomicAdd(&local_sum, sobelx[(y * pitchX) + x] - sobely[(y * pitchY) + x]);

    __syncthreads();
    if(threadIdx.x == 0 && threadIdx.y == 0) {
        x /= POOLSIZE;
        y /= POOLSIZE;
        float mean = local_sum/(POOLSIZE*POOLSIZE);
        out[x + y * pitchOut] = mean;
    }
}

void average_pooling(const uint8_t* devSobelX, const uint8_t* devSobelY, uint8_t *devOut,
                     int patchs_x, int patchs_y, int pitchX, int pitchY, int pitchOut) {

    int bsize = POOLSIZE;
    dim3 dimBlock(bsize, bsize);
    dim3 dimGrid(patchs_x, patchs_y);
    compute_avg_pooling<<<dimGrid, dimBlock>>>(devSobelX, devSobelY, devOut, patchs_x, patchs_y,
                                                pitchX, pitchY, pitchOut);
    hipDeviceSynchronize();

    if (hipPeekAtLastError())
        printf("avg pooling Error\n");

}

__global__ void dilation(const uint8_t* in, uint8_t *out, int width,
                            int height, int pitchIn, int pitchOut) {

    int r = 2;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < r || x >= width - r) return;
    if (y < r || y >= height - r) return;

    uint8_t current_val = 0;
    for (int ky = -r + 1; ky < r; ky++) { // first and last line of kernel are zeros
        for (int kx = -r; kx <= r; kx++) {
            int prop_val = in[((y + ky) * pitchIn) + (x + kx)];
            if (prop_val > current_val)
                current_val = prop_val;
        }
    }

    out[x + y * pitchOut] = current_val;
}

__global__ void erosion(const uint8_t* in, uint8_t *out, int width,
                            int height, int pitchIn, int pitchOut) {

    int r = 2;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < r || x >= width - r) return;
    if (y < r || y >= height - r) return;

    uint8_t current_val = 255;
    for (int ky = -r + 1; ky < r; ky++) {
        for (int kx = -r; kx <= r; kx++) {
            int prop_val = in[((y + ky) * pitchIn) + (x + kx)];
            if (prop_val < current_val)
                current_val = prop_val;
        }
    }

    out[x + y * pitchOut] = current_val;
}


void morph_closure(const uint8_t* devIn, uint8_t *devOut,
                    int width, int height, int pitchIn, int pitchOut) {
    hipError_t rc = hipSuccess;

    // Allocate device memory
    uint8_t* devTmp;
    size_t pitchTmp;


    rc = hipMallocPitch(&devTmp, &pitchTmp, width * sizeof(uint8_t), height);
    if (rc)
        printf("Fail tmp buffer allocation\n");

    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);

        dilation<<<dimGrid, dimBlock>>>(devIn, devTmp, width, height, pitchIn, pitchTmp);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
            printf("dilation Error\n");
        
        erosion<<<dimGrid, dimBlock>>>(devTmp, devOut, width, height, pitchTmp, pitchOut);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
            printf("erosion Error\n");
    }

    hipFree(devTmp);
}


__global__ void compute_max(const uint8_t* in, unsigned int *max,
                            int width, int height, int pitchIn) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height) return;

    atomicMax(max, in[x + y * pitchIn]);
}

__global__ void compute_threshold(const uint8_t* in,
                                    uint8_t *out, unsigned int *max, int width, int height,
                                    int pitchIn, int pitchOut) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= width || y >= height) return;
    uint8_t value = *max/2;
    out[x + y * pitchOut] = 255 * (in[x + y * pitchIn] > value);
}

void threshold(const uint8_t* devIn, uint8_t *devOut, int width, int height,
                int pitchIn, int pitchOut) {
    hipError_t rc = hipSuccess;

    // Allocate device memory
    unsigned int* devMax;

    rc = hipMalloc(&devMax, sizeof(unsigned int));
    if (rc)
        printf("Fail max variable allocation\n");

    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);

        compute_max<<<dimGrid, dimBlock>>>(devIn, devMax, width, height, pitchIn);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
            printf("max Error\n");
        
        compute_threshold<<<dimGrid, dimBlock>>>(devIn, devOut, devMax, width, height, pitchIn, pitchOut);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
            printf("thresholding Error\n");

    }
    hipFree(devMax);
}


void process_image(const uint8_t* img, uint8_t *output, int width, int height) {
    hipError_t rc = hipSuccess;

    int stride_input = width * sizeof(uint8_t);

    // Allocate device memory
    uint8_t *devImg, *devSobelX, *devSobelY;
    size_t pitchImg, pitchX, pitchY;

    rc = hipMallocPitch(&devImg, &pitchImg, width * sizeof(uint8_t), height);
    if (rc)
        printf("Fail devIn allocation\n");

    rc = hipMemcpy2D(devImg, pitchImg, img, stride_input, width, height, hipMemcpyHostToDevice);
    if (rc)
        printf("Couldn't copy img to gpu\n");


    // Sobel X & Y
    rc = hipMallocPitch(&devSobelX, &pitchX, width * sizeof(uint8_t), height);
    if (rc)
        printf("Fail devSobelX allocation\n");
    rc = hipMallocPitch(&devSobelY, &pitchY, width * sizeof(uint8_t), height);
    if (rc)
        printf("Fail devSobelY allocation\n");

    sobel_filter(devImg, devSobelX, devSobelY, width, height, pitchImg, pitchX, pitchY);

    // Average Pooling
    int new_width = std::floor((float)width / POOLSIZE);
    int new_height = std::floor((float)height / POOLSIZE);
    int stride_out = new_width * sizeof(uint8_t);

    uint8_t *devResp;
    size_t pitchResp;
    rc = hipMallocPitch(&devResp, &pitchResp, new_width * sizeof(uint8_t), new_height);
    if (rc)
        printf("Fail devIn allocation\n");
    average_pooling(devSobelX, devSobelY, devResp, new_width, new_height, pitchX, pitchY, pitchResp);

    // Morphological Closure
    uint8_t *devPostproc;
    size_t pitchPostproc;
    rc = hipMallocPitch(&devPostproc, &pitchPostproc, new_width * sizeof(uint8_t), new_height);
    if (rc)
        printf("Fail devIn allocation\n");
    morph_closure(devResp, devPostproc, new_width, new_height, pitchResp, pitchPostproc);

    // Thresholding
    uint8_t *devOutput;
    size_t pitchOutput;
    rc = hipMallocPitch(&devOutput, &pitchOutput, new_width * sizeof(uint8_t), new_height);
    if (rc)
        printf("Fail devIn allocation\n");
    threshold(devPostproc, devOutput, new_width, new_height, pitchPostproc, pitchOutput);

    // Copy back to main memory
    rc = hipMemcpy2D(output, stride_out, devOutput, pitchOutput,
                        new_width * sizeof(uint8_t), new_height, hipMemcpyDeviceToHost);
    if (rc)
        printf("Unable to copy output back to memory\n");

    hipFree(devImg);
    hipFree(devSobelX);
    hipFree(devSobelY);
    hipFree(devResp);
    hipFree(devPostproc);
    hipFree(devOutput);
}
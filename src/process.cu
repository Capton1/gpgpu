#include "hip/hip_runtime.h"
#include "process.hpp"
#include <stdio.h>

__global__ void sobel_x_filter(const unsigned char* in, float *out, int width,
                            int height, int pitch) {

    float kernel[3][3] = {{-1.0, 0.0, 1.0}, {-2.0, 0.0, 2.0}, {-1.0, 0.0, 1.0}};
    int r = 1;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < r || x >= width - r) return;
    if (y < r || y >= height - r) return;

    float sum = 0.0;
    for (int kx = -r; kx <= r; kx++) {
        for (int ky = -r; ky <= r; ky++) {
            float pixel = in[((y + ky) * pitch) + (x + kx)];
            sum += kernel[ky+r][kx+r] * pixel;
        }
    }

    out[x + y * pitch] = (sum > 0) ? sum : -sum;
}

__global__ void sobel_y_filter(const unsigned char* in, float *out, int width,
                            int height, int pitch) {

    float kernel[3][3] = {{1.0, 2.0, 1.0}, {0.0, 0.0, 0.0}, {-1.0, -2.0, -1.0}};
    int r = 1;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < r || x >= width - r) return;
    if (y < r || y >= height - r) return;

    float sum = 0.0;
    for (int kx = -r; kx <= r; kx++) {
        for (int ky = -r; ky <= r; ky++) {
            float pixel = in[((y + ky) * pitch) + (x + kx)];
            sum += kernel[ky+r][kx+r] * pixel;
        }
    }

    out[x + y * pitch] = (sum > 0) ? sum : -sum;
}

void sobel_filter(unsigned char* buffer, float *filter_output, int width, int height, int stride, char type) {
    hipError_t rc = hipSuccess;

    // Allocate device memory
    unsigned char*  devIn;
    float* devOut;
    size_t pitchIn, pitchOut;

    rc = hipMallocPitch(&devIn, &pitchIn, width * sizeof(char), height);
    if (rc)
        printf("Fail buffer allocation\n");

    rc = hipMemcpy2D(devIn, pitchIn, buffer, stride, width, height, hipMemcpyHostToDevice);
    if (rc)
        printf("Unable to copy buffer back to memory\n");

    rc = hipMallocPitch(&devOut, &pitchOut, width * sizeof(float), height);
    if (rc)
        printf("Fail buffer allocation\n");

    {
        int bsize = 32;
        int w     = std::ceil((float)width / bsize);
        int h     = std::ceil((float)height / bsize);

        dim3 dimBlock(bsize, bsize);
        dim3 dimGrid(w, h);
        if (type == 'x')
            sobel_x_filter<<<dimGrid, dimBlock>>>(devIn, devOut, width, height, pitchIn);
        else
            sobel_y_filter<<<dimGrid, dimBlock>>>(devIn, devOut, width, height, pitchIn);
        hipDeviceSynchronize();

        if (hipPeekAtLastError())
            printf("compute_filter Error\n");

    }

    // Copy back to main memory
    rc = hipMemcpy2D(filter_output, stride * sizeof(float), devOut, pitchOut,
                        width * sizeof(float), height, hipMemcpyDeviceToHost);
    if (rc)
        printf("Unable to copy buffer back to memory\n");
}